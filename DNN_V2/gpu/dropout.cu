#include "hip/hip_runtime.h"
#include <hiprand.h>
#include "dropout.h"

bool CheckDropout(const int n_layers, float const *layer_dropout) {
  bool dropout {false};
  for (int l = 0; l < n_layers; ++l) {
    if (layer_dropout[l] < 1.0f) {
      dropout = true;
      break;
    }
  }
  return dropout;
}

void RandomlySelectDropout(const int n, float *D) {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
  hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_BEST);
  hiprandSetPseudoRandomGeneratorSeed(gen, 104);
  hiprandGenerateUniform(gen, D, n);
  hiprandDestroyGenerator(gen);
}

__global__
void ApplyDropout(const int n, const float *D, float *A,
                  const float keep_prob) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    if (D[i] > keep_prob)
      A[i] = 0.0f;
    else
      A[i] /= keep_prob;
  }
}
