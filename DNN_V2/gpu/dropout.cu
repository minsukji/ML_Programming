
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

extern const int nThreads;

void RandomlySelectDropout(const int n, float *D) {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MT19937);
  hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_BEST);
  hiprandSetPseudoRandomGeneratorSeed(gen, 104);
  hiprandGenerateUniform(gen, D, n);
  hiprandDestroyGenerator(gen);
}

__global__
void ApplyDropout(const int n, const float *D, float *A,
                  const float keep_prob) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n) {
    if (D[i] > keep_prob)
      A[i] = 0.0f;
    else
      A[i] /= keep_prob;
  }
}
