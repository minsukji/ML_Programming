#include "hipblas.h"
#include "activation_func.h"
#include "dropout.h"

extern const int nThreads;

void Forward(const int n_layers, const int *layer_dims, const int batch_size,
             const float *X, const float *W, const float *B, float *Z, float *A,
             const int *W_index, const int *B_index, const int *Z_index,
             const float *oneVec, const float *layer_drop, const float *D) {
  int m, n, k, lda, ldb, ldc;
  float alpha, beta;
  int W_curLoc, B_curLoc, Z_curLoc, Z_lastLoc;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // Perform forward propagation from layer 1 to output layer L
  for (int l = 1; l <= n_layers; ++l) {
    W_curLoc = W_index[l-1];
    B_curLoc = B_index[l-1];
    Z_curLoc = Z_index[l-1];

    m = layer_dims[l];
    n = batch_size;
    k = layer_dims[l-1];
    lda = m;
    ldb = k;
    ldc = m;
    alpha = 1.0f;
    beta = 0.0f;

    // 1 Compute Z (linear)
    // 1.1 W[l] * A[l-1]
    if (l == 1)
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                  W+W_curLoc, lda, X, ldb, &beta, Z+Z_curLoc, ldc);
    else
      hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
                  W+W_curLoc, lda, A+Z_lastLoc, ldb, &beta, Z+Z_curLoc, ldc);
    // 1.2 W[l] * A[l-1] + B[l]
    hipblasSger(handle, m, n, &alpha, B+B_curLoc, 1, oneVec, 1, Z+Z_curLoc, lda);

    // 2 Compute A (non-linear)
    int nBlocks = (m*n + nThreads - 1) / nThreads;
    if (l < n_layers)
      Relu<<<nBlocks, nThreads>>>(m*n, Z+Z_curLoc, A+Z_curLoc);
    else if (l == n_layers)
      Sigmoid<<<nBlocks, nThreads>>>(m*n, Z+Z_curLoc, A+Z_curLoc);

    // 3 Modify A if dropout is applied
    if (D != nullptr && layer_drop[l] < 1.0f) {
      float keep_prob = layer_drop[l];
      int nBlocks = (m*n + nThreads - 1) / nThreads;
      ApplyDropout<<<nBlocks, nThreads>>>(m*n, D+Z_curLoc, A+Z_curLoc, keep_prob);
    }

    Z_lastLoc = Z_curLoc;
  }

  hipblasDestroy(handle);
}
