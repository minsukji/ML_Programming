#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <iostream>
#include <hiprand/hiprand.h>
#include <cmath> // for log in ComputeCostSerial
#include <random> /// for InitParamsSerial

constexpr int nThreads {64};

// Apply sigmoid function element-wise to a matrix.
__global__
void SigmoidCuda(const float *Z, const int n, float *A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        A[i] = 1.0f / (1.0f + expf(-Z[i]));
        //A[i] = fmaxf(0.0f, 1.0f / (1.0f + expf(-Z[i])));
}

// Apply rectified-linear-unit function element-wise to a matrix.
__global__
void ReluCuda(const float *Z, const int n, float *A)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        A[i] = fmaxf(Z[i], 0.0f);
}

// Compute derivative of sigmoid function element-wise for a matrix.
// For a = sigmoid(z), derivative of a with respect to z is a*(1-a).
__global__
void SigmoidBackwardCuda(const int n, const float *dA, const float *A, float *dZ)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        dZ[i] = dA[i] * (A[i] * (1.0f - A[i]));
}

// Compute derivative of relu function element-wise for a matrix.
// For a = relu(z), derivative of a with repsct to z is: 1 if z is positive, 0 otherwise.
__global__
void ReluBackwardCuda(const int n, const float *dA, const float *Z, float *dZ)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (Z[i] > 0.0f)
            dZ[i] = dA[i];
        else
            dZ[i] = 0.0f;
    }
}

__global__
void InitParams2Cuda(float *W, const int n, const int divisor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        W[i] *= sqrtf(2.0f / static_cast<float>(divisor));
}

// Randomly initialize parameters (W matrix and b vector) of each layer of DNN.
// layer_dims contains the number of activations from input to output layer.
void InitParamsCuda(const int *layer_dims, const int num_layers, const int *W_index, float *W, float *B) {
    //curandStatus_t stat;
    hiprandGenerator_t gen;
    //curandCreateGenerator(&gen, CURAND_RNG_QUASI_DEFAULT);
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 21ULL);
    //curandSetPseudoRandomGeneratorSeed(gen, 12345ULL);
    //stat = curandGenerateNormal(gen, W, W_index[num_layers], 0.0f, 1.0f);
    hiprandGenerateNormal(gen, W, W_index[num_layers], 0.0f, 1.0f);
    //if (stat == CURAND_STATUS_SUCCESS) std::cout << "Success" << '\n';
    //if (stat == CURAND_STATUS_NOT_INITIALIZED) std::cout << "Not initialized" << '\n';
    //if (stat == CURAND_STATUS_PREEXISTING_FAILURE) std::cout << "Preexisting failure" << '\n';
    //if (stat == CURAND_STATUS_LAUNCH_FAILURE) std::cout << "Launch failure" << '\n';
    //if (stat == CURAND_STATUS_LENGTH_NOT_MULTIPLE) std::cout << "Length not multiple" << '\n';
    //stat = curandGenerateNormal(gen, B, 33, 0.0f, 0.0f);
    //curandGenerateNormal(gen, B, 33, 0.0f, 0.0f);

    for (int i = 1; i <= num_layers; ++i) {
        //curandGenerateNormal(gen, W+W_index[i-1], W_index[i]-W_index[i-1], 0.0f, 1.0f);
        int nBlocks = ((W_index[i] - W_index[i-1]) + nThreads - 1) / nThreads;
        InitParams2Cuda<<<nBlocks, nThreads>>>(W+W_index[i-1], W_index[i] - W_index[i-1], layer_dims[i-1]);
    }
    hiprandDestroyGenerator(gen);
}

void InitParamsSerial(const int *layer_dims, const int num_layers, const int *W_index, float *W, float *B)
{
    float *local_W = new float[W_index[num_layers]] {};
    std::random_device rd;
    std::mt19937 e2(rd());
    //std::mt19937 e2(0);
    std::normal_distribution<> dist(0.0f, 1.0f);

    for (int l = 1; l <= num_layers; ++l) {
        int size = W_index[l] - W_index[l-1];
        int start_index = W_index[l-1];
        for (int i = start_index; i < start_index+size; ++i) {
            local_W[i] = dist(e2) / sqrt(static_cast<float>(layer_dims[l-1])); 
        }
    }

    hipMemcpy(W, local_W, W_index[num_layers] * sizeof(float), hipMemcpyHostToDevice);
    delete[] local_W;
}

__global__
void DropOutCuda(const float keep_prob, const int n, float *A, const float *D)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        if (D[i] > keep_prob)
            A[i] = 0.0f;
        else
            A[i] /= keep_prob;
    }
}

// Carry out forward propagation.
// Arguments are matrix X (input data) and params.
void ForwardCuda(const float *X, const float *W, const float *B, const float *rowVec, const int *layer_dims,
                 const float *layer_dropouts, const bool dropout, const int num_layers, const int n_samples,
                 const int *W_index, const int *B_index, const int *Z_index, float *Z, float *A, float *D)
{
    int m, n, k, lda, ldb, ldc;
    float alf = 1.0f;
    float bet = 0.0f;
    const float *alpha = &alf;
    const float *beta = &bet;
    int incx = 1, incy = 1;
    int W_curLoc, B_curLoc, Z_curLoc, Z_lasLoc;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t stat;

    // If dropout is applied, generate a random vector D that masks A vector.
    // This randomness changes at every iteration; thus generated everytime ForwardCuda is called.
    // Also, the device memory D with generated random numbers will be used by BackwardCuda.
    static int index {0};
    if (dropout) {
        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 101ULL);
        //curandSetGeneratorOffset(gen, index * Z_index[num_layers]); //OFFSET
        hiprandGenerateUniform(gen, D, Z_index[num_layers]);
        hiprandDestroyGenerator(gen);
        index++;
    }

    // Perform forward propagation from layer 1 to output layer
    for (int l = 1; l <= num_layers; ++l) {
        W_curLoc = W_index[l-1];
        B_curLoc = B_index[l-1];
        Z_curLoc = Z_index[l-1];
        //std::cout << "W_curLoc: " << W_curLoc << ", B_curLoc: " << B_curLoc << ", Z_curLoc: " << Z_curLoc << '\n';

        m = layer_dims[l];
        n = n_samples;
        k = layer_dims[l-1];
        //std::cout << "m: " << m << ", n: " << n << ", k: " << k << '\n';
        lda = m;
        ldb = k;
        ldc = m;

        // Compute Z (Linear computation)
        // 1. W[l] * A[l-1]
        if (l == 1) 
            stat=hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, W+W_curLoc, lda, X, ldb, beta, Z+Z_curLoc, ldc);
        else
            stat=hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, W+W_curLoc, lda, A+Z_lasLoc, ldb, beta, Z+Z_curLoc, ldc);
        if (stat != HIPBLAS_STATUS_SUCCESS) std::cout << "PROBLEM" << '\n';
        // 2. W[l] * A[l-1] + B[l]
        stat=hipblasSger(handle, m, n, alpha, B+B_curLoc, incx, rowVec, incy, Z+Z_curLoc, lda);
        if (stat != HIPBLAS_STATUS_SUCCESS) std::cout << "PROBLEM" << '\n';

        // Compute A (Non-linear computation)
        int nBlocks = (m*n + nThreads - 1) / nThreads;
        if (l < num_layers)
            ReluCuda<<<nBlocks, nThreads>>>(Z+Z_curLoc, m*n, A+Z_curLoc); // Relu(W * A + B)
        else if (l == num_layers)
            SigmoidCuda<<<nBlocks, nThreads>>>(Z+Z_curLoc, m*n, A+Z_curLoc); // Sigmoid(W * A + B)

        // Apply dropout regularization
        if (dropout && layer_dropouts[l] < 1.0f) {
            float keep_prob = layer_dropouts[l];
            DropOutCuda<<<nBlocks, nThreads>>>(keep_prob, m*n, A+Z_curLoc, D+Z_curLoc);
        }

        Z_lasLoc = Z_curLoc;
    }

    hipblasDestroy(handle);
}

// ComputeCostCuda2 does NOT give the right answer. For example, its results are dependent on nThreads
// Not sure if it's the shared memory issue or atomic issue or something else.
// ComputeCostCuda3 below works.
__global__
void ComputeCostCuda2(const float *AL, const int *Y, const int n_samples, float *d_cost) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i == 0)
        *d_cost = 0.0f;

    __shared__ float temp[nThreads];
    if (i < n_samples) {
        if (Y[i] == 0)
            temp[threadIdx.x] = -logf(1.0f - AL[i]);
        else if (Y[i] == 1)
            temp[threadIdx.x] = -logf(AL[i]);
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        float sum = 0.0f;
        int index = (n_samples < nThreads) ? n_samples : nThreads;
        //for (int j = 0; j < nThreads; ++j)
        for (int j = 0; j < index; ++j)
            sum += temp[j];
        sum /= static_cast<float>(n_samples);
        atomicAdd(d_cost, sum);
    }
}

__global__
void ComputeCostCuda3(const float *AL, const int *Y, const int n_samples, float *d_cost) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n_samples) {
        if (Y[i] == 0)
            d_cost[i] = -logf(1.0f - AL[i]);
        else if (Y[i] == 1)
            d_cost[i] = -logf(AL[i]);
    }
}

// Given AL (output layer activation) and Y (), compute the cost function (cross entropy).
// If L2 regularization, add the regularization component to the cost.
void ComputeCostCuda(const float *AL, const int *Y, const int n_samples, const float *W,
                     const int *W_index, const int num_layers, const float lambda,
                     float *cost, float *d_cost)
{
    int nBlocks = (n_samples + nThreads - 1) / nThreads;

    // Using ComputeCostCuda2
    //ComputeCostCuda2<<<nBlocks, nThreads>>>(AL, Y, n_samples, d_cost);
    //cudaMemcpy(cost, d_cost, sizeof(float), cudaMemcpyDeviceToHost);

    // Using ComputeCostCuda3
    ComputeCostCuda3<<<nBlocks, nThreads>>>(AL, Y, n_samples, d_cost);
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSasum(handle, n_samples, d_cost, 1, cost);
    *cost /= n_samples;

    // L2 Regularization
    if (lambda != 0.0f) {
        float *result = new float {0};
        hipblasSdot(handle, W_index[num_layers], W, 1, W, 1, result); 
        *result = *result * 0.5f * lambda / static_cast<float>(n_samples);
        *cost += *result;
    }
    hipblasDestroy(handle);
}

void ComputeCostSerial(const float *AL, const int *Y, const int n_samples, float *cost, float *d_cost)
{
    float *local_AL = new float[n_samples] {};
    int *local_Y = new int[n_samples] {};
    hipMemcpy(local_AL, AL, n_samples * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(local_Y, Y, n_samples * sizeof(float), hipMemcpyDeviceToHost);

    float sum {0.0f};
    float temp {0.0f};
    for (int i = 0; i < n_samples; ++i) {
        if (local_Y[i] == 0)
            temp = -std::log(1.0f - local_AL[i]);
        else if (local_Y[i] == 1)
            temp = -std::log(local_AL[i]);
        sum += temp;
    }

    *cost = sum / static_cast<float>(n_samples);
    delete[] local_AL; delete[] local_Y;
}

__global__
void BackwardCuda2(const int n, const int *Y, const float *A, float *dA) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        dA[i] = -static_cast<float>(Y[i]) / A[i] + (1.0f - static_cast<float>(Y[i])) / (1.0f - A[i]);
}
// Carry out backward propagation.
// Returns gradients of cost function with respect to W matrix and b vector
// In order to utilize the push_back method of vector data structure, grads
void BackwardCuda(const float *W, const float *B, const float *Z, const float *A, const float *X, const int *Y,
                  const float *oneVec, const int *layer_dims, const float *layer_dropouts, const bool dropout,
                  const int num_layers, const int n_samples, const float lambda, const int *W_index,
                  const int *B_index, const int *Z_index, float *dW, float *dB, float *dZ, float *dA, const float *D)
{
    int m, n, k, lda, ldb, ldc, n_elements;
    float alf1 = 1.0f;
    float alf2 = 1.0f / n_samples;
    float bet1 = 0.0f;
    float bet2 = lambda / n_samples;
    const float *alpha1 = &alf1;
    const float *alpha2 = &alf2;
    const float *beta1 = &bet1;
    const float *beta2 = &bet2;
    int incx = 1, incy = 1;
    int W_curLoc, B_curLoc, Z_curLoc, Z_nexLoc;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    Z_curLoc = Z_index[num_layers-1];
    n_elements = Z_index[num_layers]-Z_index[num_layers-1];

    int nBlocks = (n_elements + nThreads - 1) / nThreads;
    // Compute dA of the last layer
    BackwardCuda2<<<nBlocks, nThreads>>>(n_elements, Y, A+Z_curLoc, dA+Z_curLoc);

    int l;
    // Backpropagation from the last layer to the second layer
    for (l = num_layers; l > 1; --l) {
        W_curLoc = W_index[l-1];
        B_curLoc = B_index[l-1];
        Z_curLoc = Z_index[l-1];
        Z_nexLoc = Z_index[l-2];
        n_elements = Z_index[l] - Z_index[l-1];

        nBlocks = (n_elements + nThreads - 1) / nThreads;

        // Compute dZ
        if (l == num_layers)
            SigmoidBackwardCuda<<<nBlocks, nThreads>>>(n_elements, dA+Z_curLoc, A+Z_curLoc, dZ+Z_curLoc);
        else
            ReluBackwardCuda<<<nBlocks, nThreads>>>(n_elements, dA+Z_curLoc, Z+Z_curLoc, dZ+Z_curLoc);

        // Compute dB
        m = layer_dims[l];
        n = n_samples;
        lda = m;
        hipblasSgemv(handle, HIPBLAS_OP_N, m, n, alpha2, dZ+Z_curLoc, lda, oneVec, incx, beta1, dB+B_curLoc, incy);

        // Compute dW
        m = layer_dims[l];
        n = layer_dims[l-1];
        k = n_samples;
        lda = m;
        ldb = n;
        ldc = m;
        if (lambda != 0.0f)
            // With L2 regularization
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha2, dZ+Z_curLoc, lda,  A+Z_nexLoc, ldb, beta2, dW+W_curLoc, ldc);
        else
            // Without L2 regularization
            hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha2, dZ+Z_curLoc, lda,  A+Z_nexLoc, ldb, beta1, dW+W_curLoc, ldc);

        // Compute dA
        m = layer_dims[l-1];
        n = n_samples;
        k = layer_dims[l];
        lda = k;
        ldb = k;
        ldc = m;
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, alpha1, W+W_curLoc, lda, dZ+Z_curLoc, ldb, beta1, dA+Z_nexLoc, ldc);

        // Apply dropout regularization
        if (dropout && layer_dropouts[l-1] < 1.0f) {
            float keep_prob = layer_dropouts[l-1];
            nBlocks = (n * k + nThreads - 1) / nThreads;
            DropOutCuda<<<nBlocks, nThreads>>>(keep_prob, n*k, dA+Z_nexLoc, D+Z_nexLoc);
        }
    }

    // Backpropagation for the first Layer
    l = 1;
    W_curLoc = W_index[l-1];
    B_curLoc = B_index[l-1];
    Z_curLoc = Z_index[l-1];
    n_elements = Z_index[l] - Z_index[l-1];

    nBlocks = (n_elements + nThreads - 1) / nThreads;

    // Compute dZ
    ReluBackwardCuda<<<nBlocks, nThreads>>>(n_elements, dA+Z_curLoc, A+Z_curLoc, dZ+Z_curLoc);

    // Compute dB
    m = layer_dims[l];
    n = n_samples;
    lda = m;
    hipblasSgemv(handle, HIPBLAS_OP_N, m, n, alpha2, dZ+Z_curLoc, lda, oneVec, incx, beta1, dB+B_curLoc, incy);

    // Compute dW. Use X instead of A
    m = layer_dims[l];
    n = layer_dims[l-1];
    k = n_samples;
    lda = m;
    ldb = n;
    ldc = m;
    if (lambda != 0.0f)
        // With L2 regularization
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha2, dZ+Z_curLoc, lda,  X, ldb, beta2, dW+W_curLoc, ldc);
    else
        // Without L2 regularization
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, m, n, k, alpha2, dZ+Z_curLoc, lda,  X, ldb, beta1, dW+W_curLoc, ldc);

    hipblasDestroy(handle);
}

__global__
void UpdateParams2Cuda(const float learning_rate, const int n, const float *dW, float *W) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        W[i] -= learning_rate * dW[i];
    
}

__global__
void UpdateParams3Cuda(const float learning_rate, const int n, const float *dB, float *B) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        B[i] -= learning_rate * dB[i];
    
}

// Update parameters
void UpdateParamsCuda(const float learning_rate, const int n_W, const int n_B, const float *dW, const float *dB, float *W, float *B)
{
    int nBlocks = (n_W + nThreads - 1) / nThreads;
    UpdateParams2Cuda<<<nBlocks, nThreads>>>(learning_rate, n_W, dW, W);

    nBlocks = (n_B + nThreads - 1) / nThreads;
    UpdateParams3Cuda<<<nBlocks, nThreads>>>(learning_rate, n_B, dB, B);
}

void VectorStartIndex(const int *layer_dims, int num_layers, int m,
                      int *W_index, int *B_index, int *Z_index) {
    int W_sum {0}, B_sum {0}, Z_sum {0};

    for (int i = 0; i < num_layers; ++i) {
        W_index[i] = W_sum;
        W_sum += layer_dims[i+1] * layer_dims[i];
        B_index[i] = B_sum;
        B_sum += layer_dims[i+1];
        Z_index[i] = Z_sum;
        Z_sum += layer_dims[i+1] * m;
    }
    W_index[num_layers] = W_sum;
    B_index[num_layers] = B_sum;
    Z_index[num_layers] = Z_sum;
}
